#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {

#include "tensor_2d.h"

}

#define NUM_THREADS_PER_BLOCK 256.0


__global__
void _array_add_kernal(float *A, float *B, float *C, int n){
    //global thread id used for indexing.
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //ensure that thread is not trying to access memory outside of arrays.
    if (i<n){
        C[i] = A[i] + B[i];
    }
}

__host__
int mat_add_C(tensor_2d* mat_a, tensor_2d* mat_b, tensor_2d* mat_out){
    if (mat_a == NULL || mat_b == NULL|| mat_out == NULL){
        return 1;
    }
    else if (mat_a->n_cols != mat_b->n_cols || mat_a->n_cols != mat_out->n_cols){
        return 2;
    }
    else if (mat_a->n_rows != mat_b->n_rows || mat_a->n_rows != mat_out->n_rows){
        return 2;
    }

    unsigned int size = mat_a->n_elems*sizeof(double);

    //allocate device memory for arrays A, B, and output array.
    float *d_data_A, *d_data_B, *d_data_out;
    hipError_t err1 = hipMalloc((void**) &d_data_A, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err2 = hipMalloc((void**) &d_data_B, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err3 = hipMalloc((void**) &d_data_out, size);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3),__FILE__,__LINE__);
        return 101;
    }

    //copy contents of bariables h_A, and h_B to d_A, and d_B
    hipMemcpy(d_data_A, mat_a->data, size, hipMemcpyHostToDevice);
    hipMemcpy(d_data_B, mat_b->data, size, hipMemcpyHostToDevice);

    //kernal invocation
    _array_add_kernal<<<ceil(mat_a->n_elems/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_data_A, d_data_B, d_data_out, mat_a->n_elems);

    //copy device memory to host memory
    hipMemcpy(mat_out->data, d_data_out, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_data_A);
    hipFree(d_data_B);
    hipFree(d_data_out);

    return 0;
}

__global__
void _array_sub_kernal(float *A, float *B, float *C, int n){
    //global thread id used for indexing.
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //ensure that thread is not trying to access memory outside of arrays.
    if (i<n){
        C[i] = A[i] - B[i];
    }
}

__host__
int mat_sub_C(tensor_2d* mat_a, tensor_2d* mat_b, tensor_2d* mat_out){
    if (mat_a == NULL || mat_b == NULL|| mat_out == NULL){
        return 1;
    }
    else if (mat_a->n_cols != mat_b->n_cols || mat_a->n_cols != mat_out->n_cols){
        return 2;
    }
    else if (mat_a->n_rows != mat_b->n_rows || mat_a->n_rows != mat_out->n_rows){
        return 2;
    }

    unsigned int size = mat_a->n_elems*sizeof(double);

    //allocate device memory for arrays A, B, and output array.
    float *d_data_A, *d_data_B, *d_data_out;
    hipError_t err1 = hipMalloc((void**) &d_data_A, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err2 = hipMalloc((void**) &d_data_B, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err3 = hipMalloc((void**) &d_data_out, size);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3),__FILE__,__LINE__);
        return 101;
    }

    //copy contents of bariables h_A, and h_B to d_A, and d_B
    hipMemcpy(d_data_A, mat_a->data, size, hipMemcpyHostToDevice);
    hipMemcpy(d_data_B, mat_b->data, size, hipMemcpyHostToDevice);

    //kernal invocation
    _array_sub_kernal<<<ceil(mat_out->n_elems/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_data_A, d_data_B, d_data_out, mat_a->n_elems);

    //copy device memory to host memory
    hipMemcpy(mat_out->data, d_data_out, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_data_A);
    hipFree(d_data_B);
    hipFree(d_data_out);

    return 0;
}
__host__
int print_device_info(){
    hipDeviceProp_t dev_prop;
    for (int i = 0; i < 1; i++) {
        hipGetDeviceProperties(&dev_prop, i);

        printf("dev : %d, clockrate : %d\n", i+1, dev_prop.clockRate);
        printf("dev : %d, concurrentKernels : %d\n", i+1, dev_prop.concurrentKernels  );
        printf("dev : %d, max threads per dim : %d, %d, %d\n", i+1, dev_prop.maxThreadsDim[0],dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
        printf("dev : %d, max Threads Per Block : %d\n", i+1, dev_prop.maxThreadsPerBlock);
        printf("dev : %d, max grid siez per dim : %d, %d, %d\n", i+1, dev_prop.maxGridSize[4],dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);//futher investigation into the indecies is required.
        printf("dev : %d, warp size : %d\n", i+1, dev_prop.warpSize);
    }
    return 0;
}

__global__
void _mat_mul_kernal(float* data_a, float* data_b, float* data_out, unsigned int mat_a_num_rows, unsigned int mat_a_num_cols, unsigned int mat_b_num_cols){

    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;

    printf("row : %u, col : %u\n", row, col);

    if (row>mat_a_num_rows || col>mat_b_num_cols){
        return;
    }

    float sum = 0;
    unsigned int mat_a_index, mat_b_index;
    for (unsigned int k = 0; k<mat_a_num_cols; k++){
        mat_a_index = row*mat_a_num_cols + k;
        mat_b_index = col + k*mat_b_num_cols;
        sum += data_a[mat_a_index]*data_b[mat_b_index];
    }
    data_out[row + col*mat_b_num_cols] = sum;
}


__host__
int mat_mul_C(tensor_2d* mat_a, tensor_2d* mat_b, tensor_2d* mat_out){
        if (mat_a == NULL || mat_b == NULL || mat_out == NULL){
        return 1;
    }
    if (mat_a->n_cols != mat_b->n_rows){
        return 2;
    }
    if (mat_out->n_rows != mat_a->n_rows || mat_out->n_cols != mat_b->n_cols){
        return 2;
    }

    unsigned int size_a = mat_a->n_elems*sizeof(double);
    unsigned int size_b = mat_b->n_elems*sizeof(double);
    unsigned int size_out = mat_out->n_elems*sizeof(double);

    //allocate device memory for arrays A, B, and output array.
    float *d_data_A, *d_data_B, *d_data_out;
    hipError_t err1 = hipMalloc((void**) &d_data_A, size_a);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err2 = hipMalloc((void**) &d_data_B, size_b);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err3 = hipMalloc((void**) &d_data_out, size_out);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3),__FILE__,__LINE__);
        return 101;
    }

    //copy contents of bariables h_A, and h_B to d_A, and d_B
    hipMemcpy(d_data_A, mat_a->data, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_data_B, mat_b->data, size_b, hipMemcpyHostToDevice);

    //kernal invocation
    _mat_mul_kernal<<<ceil(mat_a->n_elems/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_data_A, d_data_B, d_data_out, mat_a->n_rows, mat_a->n_cols, mat_b->n_cols);

    //copy device memory to host memory
    hipMemcpy(mat_out->data, d_data_out, size_out, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_data_A);
    hipFree(d_data_B);
    hipFree(d_data_out);

    return 0;

}

int main(){

    print_device_info();


    tensor_2d* mat_a = mat_make(6, 6);
    tensor_2d* mat_b = mat_make(6, 6);
    tensor_2d* mat_c = mat_make(6, 6);

    mat_rand(0.0, 1.0, mat_a);
    mat_eye(mat_b);

    mat_mul_C(mat_a, mat_b, mat_c);

    mat_print(mat_a);
    mat_print(mat_b);
    mat_print(mat_c);

    // //free host memory
    mat_free(&mat_a);
    mat_free(&mat_b);
    mat_free(&mat_c);
    return 0;
}