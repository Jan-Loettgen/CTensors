#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "tensor_2d.h"

#define NUM_THREADS_PER_BLOCK 256.0


__global__
void _array_add_kernal(float *A, float *B, float *C, int n){
    //global thread id used for indexing.
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //ensure that thread is not trying to access memory outside of arrays.
    if (i<n){
        C[i] = A[i] + B[i];
    }
}

__host__
int mat_add_C(tensor_2d* mat_a, tensor_2d* mat_b, tensor_2d* mat_out){
    if (mat_a == NULL || mat_b == NULL|| mat_out == NULL){
        return 1;
    }
    else if (mat_a->n_cols != mat_b->n_cols || mat_a->n_cols != mat_out->n_cols){
        return 2;
    }
    else if (mat_a->n_rows != mat_b->n_rows || mat_a->n_rows != mat_out->n_rows){
        return 2;
    }

    unsigned int size = mat_a->n_elems*sizeof(double);

    //allocate device memory for arrays A, B, and output array.
    float *d_data_A, *d_data_B, *d_data_out;
    hipError_t err1 = hipMalloc((void**) &d_data_A, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err2 = hipMalloc((void**) &d_data_B, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err3 = hipMalloc((void**) &d_data_out, size);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3),__FILE__,__LINE__);
        return 101;
    }

    //copy contents of bariables h_A, and h_B to d_A, and d_B
    hipMemcpy(d_data_A, mat_a->data, size, hipMemcpyHostToDevice);
    hipMemcpy(d_data_B, mat_b->data, size, hipMemcpyHostToDevice);

    //kernal invocation
    _arrayAddKernal<<<ceil(mat_a->n_elems/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_data_A, d_data_B, d_data_out, mat_a->n_elems);

    //copy device memory to host memory
    hipMemcpy(mat_out->data, d_data_out, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_data_A);
    hipFree(d_data_B);
    hipFree(d_data_out);

    return 0;
}

__global__
void _array_sub_kernal(float *A, float *B, float *C, int n){
    //global thread id used for indexing.
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //ensure that thread is not trying to access memory outside of arrays.
    if (i<n){
        C[i] = A[i] - B[i];
    }
}

__host__
int mat_sub_C(tensor_2d* mat_a, tensor_2d* mat_b, tensor_2d* mat_out){
    if (mat_a == NULL || mat_b == NULL|| mat_out == NULL){
        return 1;
    }
    else if (mat_a->n_cols != mat_b->n_cols || mat_a->n_cols != mat_out->n_cols){
        return 2;
    }
    else if (mat_a->n_rows != mat_b->n_rows || mat_a->n_rows != mat_out->n_rows){
        return 2;
    }

    unsigned int size = mat_a->n_elems*sizeof(double);

    //allocate device memory for arrays A, B, and output array.
    float *d_data_A, *d_data_B, *d_data_out;
    hipError_t err1 = hipMalloc((void**) &d_data_A, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err2 = hipMalloc((void**) &d_data_B, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2),__FILE__,__LINE__);
        return 101;
    }

    hipError_t err3 = hipMalloc((void**) &d_data_out, size);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3),__FILE__,__LINE__);
        return 101;
    }

    //copy contents of bariables h_A, and h_B to d_A, and d_B
    hipMemcpy(d_data_A, mat_a->data, size, hipMemcpyHostToDevice);
    hipMemcpy(d_data_B, mat_b->data, size, hipMemcpyHostToDevice);

    //kernal invocation
    _array_sub_kernal<<<ceil(mat_a->n_elems/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_data_A, d_data_B, d_data_out, mat_a->n_elems);

    //copy device memory to host memory
    hipMemcpy(mat_out->data, d_data_out, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_data_A);
    hipFree(d_data_B);
    hipFree(d_data_out);

    return 0;
}
__host__
int print_device_info(){
    hipDeviceProp_t dev_prop;
    for (int i = 0; i < dev_count; i++) {
        hipGetDeviceProperties(&dev_prop, i);

        printf("dev : %d, clockrate : %d\n", i+1, dev_prop.clockRate);
        printf("dev : %d, concurrentKernels : %d\n", i+1, dev_prop.concurrentKernels  );
        printf("dev : %d, max threads per dim : %d, %d, %d\n", i+1, dev_prop.maxThreadsDim[0],dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
        printf("dev : %d, max Threads Per Block : %d\n", i+1, dev_prop.maxThreadsPerBlock);
        printf("dev : %d, max grid siez per dim : %d, %d, %d\n", i+1, dev_prop.maxGridSize[0],dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
        printf("dev : %d, warp size : %d\n", i+1, dev_prop.warpSize);
    }
    return 0;
}


int main(){




    tensor_2d* mat_a = mat_make(24, 24);
    tensor_2d* mat_b = mat_make(24, 24);
    tensor_2d* mat_c = mat_make(24, 24);

    mat_rand(0.0, 1.0, mat_a);
    mat_zeros(mat_b);

    mat_add_C(mat_a, mat_b, mat_c);

    mat_print(mat_a);
    mat_print(mat_b);
    mat_print(mat_c);

    //free host memory
    mat_free(&mat_a);
    mat_free(&mat_b);
    mat_free(&mat_c);
    return 0;
}