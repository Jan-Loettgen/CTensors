#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#define NUM_THREADS_PER_BLOCK 256.0


__global__
void vecAddKernal(float *A, float *B, float *C, int n){
    //global thread id used for indexing.
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //ensure that thread is not trying to access memory outside of arrays.
    if (i<n){
        C[i] = A[i] + B[i];
    }
}

__host__
int array_add_C(float *h_A, float *h_B, float *h_C, int num_elems, int size){
    //allocate device memory for arrays A, B, and C.
    float *d_A, *d_B, *d_C;
    hipError_t err1 = hipMalloc((void**) &d_A, size);
    if (err1 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err1),__FILE__,__LINE__);
        return 11;
    }

    hipError_t err2 = hipMalloc((void**) &d_B, size);
    if (err2 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err2),__FILE__,__LINE__);
        return 11;
    }

    hipError_t err3 = hipMalloc((void**) &d_C, size);
    if (err3 != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err3),__FILE__,__LINE__);
        return 11;
    }

    //copy contents of bariables h_A, and h_B to d_A, and d_B
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //kernal invocation
    vecAddKernal<<<ceil(num_elems/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_A, d_B, d_C, num_elems);

    //copy device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}


int main(){
    int num_elems = 24;
    unsigned int size = num_elems*sizeof(float);

    //allocate host memory for arrays A, B, and C.
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    ///set arrays to contain random values.
    for (int i = 0; i< 10; i++){
        h_A[i] = rand()/2147483647.0;
    }

    for (int i = 0; i< 10; i++){
        h_B[i] = rand()/2147483647.0;
    }

    for (int i = 0; i< 10; i++){
        h_C[i] = rand()/2147483647.0;
    }


    array_add_C(h_A, h_B, h_C, num_elems, size);


    //do stuff with final array
    for (int i = 0; i< 10; i++){
        printf("%f ", h_A[i]);
    }
    printf("\n");
    for (int i = 0; i< 10; i++){
        printf("%f ", h_B[i]);
    }
    printf("\n");
    for (int i = 0; i< 10; i++){
        printf("%f ", h_C[i]);
    }
    printf("\n");

    //free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}